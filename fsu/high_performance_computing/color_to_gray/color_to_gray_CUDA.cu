#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "../Jpegfile.h"


__global__ void luminanceCalc(BYTE *dataBufD, int ND){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int i=index; i<ND; i+=stride){    
        BYTE *pRed, *pGrn, *pBlu;
        pRed = dataBufD + i*3;
        pGrn = dataBufD + i*3 +  1;
        pBlu = dataBufD + i*3 +  2;

        int lum = (int)(.299 * (double)(*pRed) + .587 * (double)(*pGrn) + .114 * (double)(*pBlu));

        *pRed = (BYTE)lum;
        *pGrn = (BYTE)lum;
        *pBlu = (BYTE)lum;
    }
}

// #define blocks 1
// #define threadsPerBlock 1
int main(){

    int experiment = 1;

    for(int blockMult=0; blockMult<experiment; blockMult++){
        printf("%d & ", (int)(pow(2, blockMult)));
        for(int threadMult=0; threadMult<experiment; threadMult++){

            clock_t start, end;
            double cpu_time_used;
             
            start = clock();

            int blocks = (int)(pow(2, blockMult)), threadsPerBlock = (int)(pow(2, threadMult));

            UINT height, width;
            BYTE *dataBuf, *dataBufD;

            dataBuf = JpegFile::JpegFileToRGB("images/monarchColor.jpg", &width, &height);

            int sizeDataBuf = width*height*3 * (int)sizeof(BYTE);

            int N=height*width;

            // Allocating Space for device copies
            hipMalloc((void **)&dataBufD, sizeDataBuf);

            // Copy inputs into the device
            hipMemcpy(dataBufD, dataBuf, sizeDataBuf, hipMemcpyHostToDevice);

            // Launch the luminanceCalc function
            luminanceCalc<<<blocks,threadsPerBlock>>>(dataBufD, N);

            // Wait for GPU to finish before accessing on host
            hipDeviceSynchronize();

            // Copy new data from the device
            hipMemcpy(dataBuf, dataBufD, sizeDataBuf, hipMemcpyDeviceToHost);

            // Write the gray luminance to another jpg file
            JpegFile::RGBToJpegFile("images/monarchMonoExp.jpg", dataBuf, width, height, true, 75);
            
            // Cleanup 
            free(dataBuf);
            hipFree(dataBufD);

            end = clock();
            cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
            printf("%f & ", cpu_time_used);

        }
        printf("\\\\ \n");
    }

    return 0;
}

