#include "hip/hip_runtime.h"
// #ifdef DOUBLE
typedef double precision;
// #else
// typedef float precision;
// #endif

#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "../Jpegfile.h"

            // averageCalc<<<blocks, threadsPerBlock>>>(dataBufD, dataAvD, distancesD, distancesRowStartIndexD, rowStartD, rowEndD, N, height, width, sum, rInt);

__global__ void averageCalc(BYTE *dataBufD, BYTE *dataAvD, precision *distancesD, int *distancesRowStartIndexD, int *rowStartD, int *rowEndD, UINT height, UINT width, int rInt){

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
    int const rInt21 = rInt*2+1;
    int ND = (int)(height*width);

	for(int pixel=index; pixel<ND; pixel+=stride){
		// These are for the borders and get the row and col of the pixel this thread is going to handle
		int borderTop = 0, borderBot = 0, row = (int)(pixel/(int)width);
		int col = pixel - row*(int)width;
		// int *rowStartBord = (int *)malloc(rInt21*sizeof(int)), *rowEndBord = (int *)malloc(rInt21*sizeof(int));
		int rowStartBord[11], rowEndBord[11];
		for(int i=0; i<rInt21; i++){
			rowStartBord[i] = rowStartD[i];
			rowEndBord[i] = rowEndD[i];
		}

		// ////// These then change the border variables accordingly if we need to //////
		// // This tell the next nested loop to start at a higher row 
		// if(row < rInt){ borderTop = rInt - row; }

		// // This will tell the next nested loop to stop at a lower row
		// if(row > (int)height - rInt - 1){ borderBot = row - ((int)height - rInt - 1); }

		// // This is for the left boundary and changes the indices of rowStartBord if they are outside boundary
		// if(col < rInt){		
		// 	for(int i=0; i< rInt21; i++){
		// 		if(rowStartD[i] < -1*col){
		// 			rowStartBord[i] = -1*col;
		// 		}
		// 	}
		// }
		// // This is for the right boundary and changes the indices of rowEndBord if the end outside boundary
		// if(col > (int)width - rInt - 1){		
		// 	for(int i=0; i<rInt21; i++){
		// 		if(rowEndD[i] > (int)width - col - 1){
		// 			rowEndBord[i] = (int)width - col - 1;
		// 		}
		// 	}
		// }

		// rowStartD[0] = borderTop;
		// rowEndD[0] = borderBot;
		// for(int i=1; i<rInt21; i++){
		// 	rowEndD[i] = rowEndBord[i];
		// 	rowStartD[i] = rowStartBord[i];
		// }

		// These will hold the averages of the three colors
		precision rAvg = 0.0, gAvg = 0.0, bAvg = 0.0;

		BYTE *pRed, *pGrn, *pBlu;
		precision sum = 0.0;
		for(int i=borderTop; i <= rInt*2 - borderBot; i++){
			for(int j=0; j <= rowEndBord[i] - rowStartBord[i]; j++){
				if( (row+i-rInt) * (int)width + (col+rowStartBord[i]+j) >= 0 && (row+i-rInt) * (int)width + (col+rowStartBord[i]+j) < ND){
					// We need to shift the row and columns accordingly to the pixel in the ball we want
					// row is at center of ball then (i-rInt) is the +-indicating if we are above the center(-) or below(+)
					// col is at center of ball and rowStartBord[i] is 0 at most so j runs throught the row of the ball
					pRed = dataBufD + (row+i-rInt) * (int)width * 3 + (col+rowStartBord[i]+j) * 3;
					pGrn = dataBufD + (row+i-rInt) * (int)width * 3 + (col+rowStartBord[i]+j) * 3 + 1;
					pBlu = dataBufD + (row+i-rInt) * (int)width * 3 + (col+rowStartBord[i]+j) * 3 + 2;

					// This adds new information to the average multiplied by the corresponding weight 
					// distanceRowStartIndex starts you at the first spot of the this row on the stencil, 
						// but if we are on the left border we need to start not on the left most but a however many to the right which is what rowStartBord - rowStart does,
						// then j just is the increment for moving through the row 
					precision distance = distancesD[distancesRowStartIndexD[i] + rowStartBord[i]-rowStartD[i] + j];
					sum += distance;
					rAvg += ((precision)(*pRed))*distance;
					gAvg += ((precision)(*pGrn))*distance;
					bAvg += ((precision)(*pBlu))*distance;
				}
			}	
		}
		// rowStartD[0] = (int)(rAvg/sum), rowStartD[1] = (int)(gAvg/sum), rowStartD[2] = (int)(bAvg/sum), rowStartD[3] = (int)(sum*100000);

		pRed = dataAvD + pixel*3, pGrn = dataAvD + pixel*3 + 1, pBlu = dataAvD + pixel*3 + 2;

		// Divide by the sum to get the weighted average and turn it into an int and add it to the vector
        *pRed = (BYTE)(rAvg/sum), *pGrn = (BYTE)(gAvg/sum), *pBlu = (BYTE)(bAvg/sum);

        free(rowStartBord);
        free(rowEndBord);
	}

}


int main(){

	int experiment = 8;

    for(int blockMult=1; blockMult<experiment; blockMult++){
        printf("%d & ", (int)(pow(2, blockMult)));
        for(int threadMult=1; threadMult<experiment; threadMult++){

            clock_t start, end;
            double cpu_time_used;
             
            start = clock();

            int blocks = (int)(pow(2, blockMult)), threadsPerBlock = (int)(pow(2, threadMult));

			precision pi = M_PI;

			precision radius = 5.0;
			precision std = 10.0;
			precision std2 = 2*std*std;
			precision bottom = pi*std2;		// what the exponential function gets divided by

			// This is pre-processing to get distances/weights of gaussian distribution into a vector of vectors
			// We will find the border of our ball and then calculate the distances of each pixel in our ball from our zero pixel
			// In this we will treat the center of the ball as the the origin so roww=0 and coll=0. 
			// RowStart and RowEnd are inclusive so rowStart[2] = -2 and rowEnd[2] = 2; the row has 5 elements with index (-2,-1,0,1,2)
			int rInt = (int)radius;
			int rInt21 = rInt*2+1;
			int *rowStart = (int *)malloc(rInt21*sizeof(int)), *rowStartD;
			int *rowEnd = (int *)malloc(rInt21*sizeof(int)), *rowEndD;
			int roww = -1*rInt;		// This is the row of the top most pixel of ball
			int coll = 0;			// This is the column of the top most pixel of ball
			precision *distances, *distancesD;

			// printf("Done Allocating for rowStart and rowEnd, rInt = %d\n", rInt);

			// This gets the 2nd qudrant of the border and we will use symmetry to get the rest of the border
			// This starts at top and finds the border by moving left measuring the distance from center and
			// 		if the distance is larger than the radius we move back right, put that index into the our rowStart indices vector, and move down to start this over
			//		if it is smaller then we start the while loop over again to find the start of this row
			while(roww != 0){	
				coll--;
				precision length = sqrt(coll*coll + roww*roww);
				if(length > radius){
					coll++;
					rowStart[rInt + roww] = coll;
					roww++;
				}
			}
			rowStart[rInt] = -1*rInt; // This is the middle row start

			//// These next two for loops fill in rowStart and rowEnd using symmetry ////
			// the +-i from center will have same starting point
			for(int i=1; i <= rInt; i++){	rowStart[rInt+i] = rowStart[rInt-i];   }
			// the ends are same distance from center but on other side so just times them by negative 1
			for(int i=0; i < rInt21; i++){	rowEnd[i] = -1*rowStart[i];	  }
			// for(int i=0; i < rInt21; i++){	printf("%d, ", rowEnd[i]);	  }
			// printf("\n");
			// This frees up memory in the GPU and copies over rowStart and rowEnd into that freed memory
			hipMalloc((void **)&rowStartD, rInt21*sizeof(int));
			hipMalloc((void **)&rowEndD, rInt21*sizeof(int));
			hipMemcpy(rowStartD, rowStart, rInt21*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(rowEndD, rowEnd, rInt21*sizeof(int), hipMemcpyHostToDevice);

			// printf("Done Allocating and Copying to rowStartD and rowEndD\n");

			// This allocates memory for distances and *distancesRowStartIndex
			int distancesSize = 0;
			int *distancesRowStartIndex = (int *)malloc(rInt21*sizeof(int)), *distancesRowStartIndexD;
			for(int i=0; i<(rInt*2+1); i++){
				distancesRowStartIndex[i] = distancesSize;
				distancesSize += rowEnd[i]*2+1;
			}
			distances = (precision *)malloc((distancesSize)*sizeof(precision));

			// printf("Done Allocating memory for distances and distancesRowStartIndex\n");

			// This fills in the distances data structure by just going through the the rows of the ball and calculating the distance and storing it
				// Fix sum if I have time
			precision sum = 0.0;
			for(int i=0; i<=rInt*2; i++){
				for(int j=0; j<rowEnd[i]-rowStart[i]; j++){			  // This underneat is the actual distance from center
					distances[distancesRowStartIndex[i]+j] = exp(-1 * sqrt( (rInt-i)*(rInt-i) + (rowStart[i]+j)*(rowStart[i]+j) ) / std2) / bottom;
					sum = sum + distances[distancesRowStartIndex[i]+j];
				}
			}
			// printf("sum = %f\n", sum);

			// printf("Done calculating Distances (Weights)\n");

			// This frees up memory in the GPU and copies over distances and distancesRowStartIndex
			hipMalloc((void **)&distancesRowStartIndexD, rInt21*sizeof(int));
			hipMalloc((void **)&distancesD, distancesSize*sizeof(precision));
			hipMemcpy(distancesRowStartIndexD, distancesRowStartIndex, rInt21*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(distancesD, distances, distancesSize*sizeof(precision), hipMemcpyHostToDevice);

			// printf("Copied distances and distancesRowStartIndex in Device Memory\n");

			UINT height, width;
            BYTE *dataBuf, *dataBufD, *dataAvD;

            dataBuf = JpegFile::JpegFileToRGB("images/monarchColor.jpg", &width, &height);

            // printf("Height = %d\n", (int)height);
            // printf("Width = %d\n", (int)width);

            int sizeDataBuf = width*height*3 * (int)sizeof(BYTE);
            int N=height*width;

            // Allocating Space for device copies and copyies them into device
            BYTE *dataAv = (BYTE *)malloc(sizeDataBuf);
            for(int i=0; i<3*N; i++){    dataAv[i] = 0;    }
            hipMalloc((void **)&dataBufD, sizeDataBuf);
            hipMalloc((void **)&dataAvD, sizeDataBuf);
            hipMemcpy(dataBufD, dataBuf, sizeDataBuf, hipMemcpyHostToDevice);
            hipMemcpy(dataAvD, dataAv, sizeDataBuf, hipMemcpyHostToDevice);

            // printf("Copied dataBuf and dataAv in Device Memory\n");

            averageCalc<<<blocks, threadsPerBlock>>>(dataBufD, dataAvD, distancesD, distancesRowStartIndexD, rowStartD, rowEndD, height, width, rInt);

            // printf("Finished Kernal\n");

            // Wait for GPU to finish before accessing on host
            hipDeviceSynchronize();

            // Copy new data from the device
            hipMemcpy(dataBuf, dataAvD, sizeDataBuf, hipMemcpyDeviceToHost);
            hipMemcpy(rowStart, rowStartD, rInt21*sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(rowEnd, rowEndD, rInt21*sizeof(int), hipMemcpyDeviceToHost);
            // for(int i=0; i<rInt21; i++){	printf("%d, ", rowStart[i]);	}
           	// printf("\n");
            // for(int i=0; i < rInt21; i++){	printf("%d, ", rowEnd[i]);	}
            // printf("\n");
            // printf("%d, %d, %d \n", dataBuf[100*227*3], dataBuf[100*227*3+1], dataBuf[100*227*3+2]);

			// Write the gray lumin to another jpg file
            JpegFile::RGBToJpegFile("images/monarchBlur5.jpg", dataBuf, width, height, true, 100);

            // printf("Finished Jpeg Image\n");
            
            // Cleanup 
            free(dataBuf);
            hipFree(dataBufD);
 			hipFree(dataAvD);

 			free(distances);
 			hipFree(distancesD);
 			free(distancesRowStartIndex);
 			hipFree(distancesRowStartIndexD);

 			free(rowStart);
 			hipFree(rowStartD);
 			free(rowEnd);
 			hipFree(rowEndD);

 			// printf("Finished Cleanup\n");

            end = clock();
            cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
            printf("%f & ", cpu_time_used);

        }
        printf("\\\\ \n");
    }

    return 0;
}

